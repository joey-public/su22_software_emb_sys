#include "hip/hip_runtime.h"
#include "img_proc.h"

// =================== Helper Functions ===================
inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

// =================== CPU Functions ===================

void img_rgb2gray_cpu(uchar* out, const uchar* in, const uint width, const uint height, const int channels)
{
    int out_idx = 0;
    int avg = 0;
    for(int i = 0; i < channels*width*height; i++){
        avg += in[i];
        if((i+1)%channels == 0){
            avg = avg / channels;
            out[out_idx] = avg; 
            avg = 0;
            out_idx++; 
        }     
    }
}

void img_invert_cpu(uchar* out, const uchar* in, const uint width, const uint height)
{
    //TODO: Invert a 8bit image
}

void img_blur_cpu(uchar* out, const uchar* in, const uint width, const uint height, const int blur_size)
{
    //TODO: Average out blur_size pixels
}

// =================== GPU Kernel Functions ===================
/*
TODO: Write GPU kernel functions for the above functions
   */
__global__ void kernel_img_rgb2gray(uchar* out, const uchar* in, const uint width, const uint height, const int channels)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; 
    const int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int out_idx, in_idx;
    if(x < width && y < height){
        out_idx = y*width + x;
        in_idx = out_idx * channels;
        uchar r = in[in_idx];
        uchar g = in[in_idx+1];
        uchar b = in[in_idx+2];
        uchar gray_pixel = (r+g+b)/channels;
        out[out_idx] = gray_pixel;
    }
}

void kernel_img_invert(uchar* out, const uchar* in, const uint width, const uint height)
{
}

void kernel_img_blur(uchar* out, const uchar* in, const uint width, const uint height, const int blur_size)
{
}

// =================== GPU Host Functions ===================
/* 
TODO: Write GPU host functions that launch the kernel functions above
   */
void img_rgb2gray(uchar* out, const uchar* in, const uint width, const uint height, const int channels)
{
    const int grid_x = 64;
    const int grid_y = 64;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(divup(width, grid_x), divup(height, grid_y), 1);
    kernel_img_rgb2gray<<<grid,block>>>(out, in, width, height, channels);
    hipDeviceSynchronize(); 
}

void img_invert(uchar* out, const uchar* in, const uint width, const uint height)
{
}

void img_blur(uchar* out, const uchar* in, const uint width, const uint height, const int blur_size)
{
}
