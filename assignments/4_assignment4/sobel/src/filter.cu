#include "hip/hip_runtime.h"
#include "filter.h"
#include "timer.h"

#include <iostream>

#define KERNEL_SZ 3
const int sobel_kernel_x[KERNEL_SZ][KERNEL_SZ] = {
	{ 1,  0, -1},
	{ 2,  0, -2},
	{ 1,  0, -1}};
const int sobel_kernel_y[KERNEL_SZ][KERNEL_SZ] = {
	{ 1, 2, 1},
	{ 0, 0, 0},
	{-1, -2, -1}};

using namespace std;

// =================== Helper Functions ===================
inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

// =================== CPU Functions ===================
void sobel_filter_cpu(const uchar * input, uchar * output, const uint height, const uint width)
{
    float result_x, result_y, ac_x, ac_y;
    int idx;
    int offset = KERNEL_SZ/2;
    for(int i = 0; i < width*height; i++){
        result_x = 0.0f; result_y = 0.0f;
        idx = (i-width*offset)-offset;
        for(int r = 0; r < KERNEL_SZ; r++){
            for(int c = 0; c < KERNEL_SZ; c++){
                if(idx < 0 || idx > width*height){
                    ac_x = 0.0f; ac_y = 0.0f;
                }
                else{
                    ac_x = (float) input[idx] * (float) sobel_kernel_x[r][c];
                    ac_y = (float) input[idx] * (float) sobel_kernel_y[r][c];
                }
                result_x += ac_x;
                result_y += ac_y;
                idx += 1;                 
            }
            idx -= (KERNEL_SZ-1);
            idx += width;
        }
//        output[i] = (uchar) sqrt((result_x*result_x) + (result_y*result_y));
        output[i] = (uchar) ((result_x/2) + (result_y/2));
    }
}

// =================== GPU Kernel Functions ===================
__global__ void kernel_sobel_filter(const uchar * input, uchar * output, const uint height, const uint width)
{
    const int sobel_kernel_x[KERNEL_SZ][KERNEL_SZ] = {
    	{ 1,  0, -1},
    	{ 2,  0, -2},
    	{ 1,  0, -1}};
    const int sobel_kernel_y[KERNEL_SZ][KERNEL_SZ] = {
    	{ 1, 2, 1},
    	{ 0, 0, 0},
    	{-1, -2, -1}};
    const int x = blockIdx.x * blockDim.x + threadIdx.x; 
    const int y = blockIdx.y * blockDim.y + threadIdx.y; 
    const int offset = KERNEL_SZ/2;
    float result_x, result_y, ac_x, ac_y;
    int idx, i;
    if(x < width && y < height){
        i = y*width + x;
        result_x=0; 
        result_y=0;
        ac_x=0;
        ac_y=0;
        idx = (i-width*offset)-offset;
        for(int r = 0; r < KERNEL_SZ; r++){
            for(int c = 0; c < KERNEL_SZ; c++){
                if(idx < 0 || idx > width*height){
                    ac_x = 0.0f; ac_y = 0.0f;
                }
                else{
                    ac_x = (float) input[idx] * (float) sobel_kernel_x[r][c];
                    ac_y = (float) input[idx] * (float) sobel_kernel_y[r][c];
                }
                result_x += ac_x;
                result_y += ac_y;
                idx += 1;                 
            }
            idx -= (KERNEL_SZ-1);
            idx += width;
        }
        output[i] = (uchar) ((result_x/2) + (result_y/2));
    }
}
// =================== GPU Host Functions ===================
void sobel_filter_gpu(const uchar * input, uchar * output, const uint height, const uint width)
{
    const int grid_x = 64;
    const int grid_y = 64;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(divup(width, grid_x), divup(height, grid_y), 1);
    kernel_sobel_filter<<<grid,block>>>(input, output, height, width);
    hipDeviceSynchronize(); 
}
