#include "hip/hip_runtime.h"
#include "filter.h"
#include "timer.h"

#include <iostream>

#define KERNEL_SZ 3
int sobel_kernel_x[KERNEL_SZ][KERNEL_SZ] = {
	{ 1,  0, -1},
	{ 2,  0, -2},
	{ 1,  0, -1}};
int sobel_kernel_y[KERNEL_SZ][KERNEL_SZ] = {
	{ 1, 2, 1},
	{ 0, 0, 0},
	{-1, -2, -1}};

using namespace std;

// =================== Helper Functions ===================
inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

// =================== CPU Functions ===================
void sobel_filter_cpu(const uchar * input, uchar * output, const uint height, const uint width)
{
    float result_x, result_y, ac_x, ac_y;
    int idx;
    int offset = KERNEL_SZ/2;
    for(int i = 0; i < width*height; i++){
        result_x = 0.0f; result_y = 0.0f;
        idx = (i-width*offset)-offset;
        for(int r = 0; r < KERNEL_SZ; r++){
            for(int c = 0; c < KERNEL_SZ; c++){
                if(idx < 0 || idx > width*height){
                    ac_x = 0.0f; ac_y = 0.0f;
                }
                else{
                    ac_x = (float) input[idx] * (float) sobel_kernel_x[r][c];
                    ac_y = (float) input[idx] * (float) sobel_kernel_y[r][c];
                }
                result_x += ac_x;
                result_y += ac_y;
                idx += 1;                 
            }
            idx -= (KERNEL_SZ-1);
            idx += width;
        }
//        output[i] = (uchar) sqrt((result_x*result_x) + (result_y*result_y));
        output[i] = (uchar) ((result_x/2) + (result_y/2));
    }
}

// =================== GPU Kernel Functions ===================

// =================== GPU Host Functions ===================
void sobel_filter_gpu(const uchar * input, uchar * output, const uint height, const uint width)
{
}
