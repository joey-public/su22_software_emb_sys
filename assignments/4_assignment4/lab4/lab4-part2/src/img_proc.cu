#include "hip/hip_runtime.h"
#include "img_proc.h"
#include <cstdio> 

// =================== Helper Functions ===================
inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

// =================== CPU Functions ===================

void img_rgb2gray_cpu(uchar* out, const uchar* in, const uint width, const uint height, const int channels)
{
    int out_idx = 0;
    int avg = 0;
    for(int i = 0; i < channels*width*height; i++){
        avg += in[i];
        if((i+1)%channels == 0){
            avg = avg / channels;
            out[out_idx] = avg; 
            avg = 0;
            out_idx++; 
        }     
    }
}

void img_invert_cpu(uchar* out, const uchar* in, const uint width, const uint height)
{
    for(int i = 0; i < width*height; i++){
        out[i] = 255-in[i];     
    }     
    
}

void img_blur_cpu(uchar* out, const uchar* in, const uint width, const uint height, const int blur_size)
{
    float scale = 1/blur_size;
    int offset = blur_size/2;
    for(int i = 0; i < width*height; i++){
        int result = 0;
        int ac = 0;
        int idx = (i-width*offset)-offset;
        for(int j = 0; j<blur_size; j++){
            for(int j =0; j<blur_size; j++){
                if(idx < 0 || idx > width*height){
                    ac = 0;
                }
                else{
                    ac = in[idx];
                }
                result += ac;
                idx+=1;
            }
            idx -= (blur_size-1);
            idx += width;
        }
        out[i] = result / (blur_size*blur_size);
    }
}

// =================== GPU Kernel Functions ===================
/*
TODO: Write GPU kernel functions for the above functions
   */
__global__ void kernel_img_rgb2gray(uchar* out, const uchar* in, const uint width, const uint height, const int channels)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; 
    const int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int out_idx, in_idx;
    if(x < width && y < height){
        out_idx = y*width + x;
        in_idx = out_idx * channels;
        uchar r = in[in_idx];
        uchar g = in[in_idx+1];
        uchar b = in[in_idx+2];
        uchar gray_pixel = (r+g+b)/channels;
        out[out_idx] = gray_pixel;
    }
}

__global__ void kernel_img_invert(uchar* out, const uchar* in, const uint width, const uint height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; 
    const int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int idx;
    if(x < width && y < height){
        idx = y*width + x;
        out[idx] = 255-in[idx];
    }
}

__global__ void kernel_img_blur(uchar* out, const uchar* in, const uint width, const uint height, const int blur_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; 
    const int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int offset = blur_size/2;
    int i;
    if(x < width && y < height){
        i = y*width + x;
        int result = 0;
        int ac = 0;
        int idx = (i-width*offset)-offset;
        for(int j = 0; j<blur_size; j++){
            for(int j =0; j<blur_size; j++){
                if(idx < 0 || idx > width*height){
                    ac = 0;
                }
                else{
                    ac = in[idx];
                }
                result += ac;
                idx+=1;
            }
            idx -= (blur_size-1);
            idx += width;
        }
        out[i] = result / (blur_size*blur_size);
    }
}

// =================== GPU Host Functions ===================
/* 
TODO: Write GPU host functions that launch the kernel functions above
   */
void img_rgb2gray(uchar* out, const uchar* in, const uint width, const uint height, const int channels)
{
    const int grid_x = 64;
    const int grid_y = 64;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(divup(width, grid_x), divup(height, grid_y), 1);
    kernel_img_rgb2gray<<<grid,block>>>(out, in, width, height, channels);
    hipDeviceSynchronize(); 
}

void img_invert(uchar* out, const uchar* in, const uint width, const uint height)
{
    const int grid_x = 64;
    const int grid_y = 64;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(divup(width, grid_x), divup(height, grid_y), 1);
    kernel_img_invert<<<grid,block>>>(out, in, width, height);
    hipDeviceSynchronize(); 
}

void img_blur(uchar* out, const uchar* in, const uint width, const uint height, const int blur_size)
{
    const int grid_x = 64;
    const int grid_y = 64;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(divup(width, grid_x), divup(height, grid_y), 1);
    kernel_img_blur<<<grid,block>>>(out, in, width, height, blur_size);
    hipDeviceSynchronize(); 
}
