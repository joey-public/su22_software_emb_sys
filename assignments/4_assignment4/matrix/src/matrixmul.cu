#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "matrixmul.h"
#include "timer.h"

#define BLOCK_SIZE 2 

__global__ void block_mm_kernel(const float* A, const float* B, float* output, int M, int N, const int block_size) 
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x>N || y>N) { return; }
    int a_idx = y*N;
    int b_idx = x;
    int c_idx = y*N+x;
    output[c_idx] = 0.0f;
    for(int i=0; i<N/blockDim.x; i++)
    {
        output[c_idx] += A[a_idx]*B[b_idx];
        a_idx += 1;
        b_idx += N;
    }
    __syncthreads();
}

inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

float run_mm_gpu(const float* A, const float* B, float* C, int M, int N)
{
	Timer gpu_timer;
	gpu_timer.start();
    dim3 blocks(N/BLOCK_SIZE,N/BLOCK_SIZE,1);
    dim3 threads(BLOCK_SIZE,BLOCK_SIZE,1);
    block_mm_kernel<<<blocks,threads>>>(A,B,C,M,N,BLOCK_SIZE);
	CudaCheckError();
	CudaSafeCall(hipDeviceSynchronize());
	gpu_timer.stop();
	float gpu_time = gpu_timer.getElapsed();
	gpu_timer.end();
	return gpu_time;
}
