#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "matrixmul.h"
#include "timer.h"

#define BLOCK_SIZE 16

__global__ void block_mm_kernel(const float* A, const float* B, float* output, int M, int N, const int block_size) 
{
    const int block = blockIdx.x;
    const int thread = threadIdx.x;
    int idx_a, idx_b, idx_out;
    idx_out = block*block_size + thread;
    idx_a = (blockDim.x / block)*block_size*M;
    idx_b = (blockDim.x / block)*block_size*N;
    output[idx_out]=0;
    for(int i = 0; i<block_size; i++){//bs # of dot products
        for(int j=0; j<block_size; j++){//do the dot product
            output[idx_out] += A[idx_a]*B[idx_b]; 
            idx_a += 1;
            idx_a += block_size;
        }
        idx_a += 1;
        idx_b += N;
    }
}


inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}


float run_mm_gpu(const float* A, const float* B, float* C, int M, int N)
{
	Timer gpu_timer;
	gpu_timer.start();
    int blocks = (N/BLOCK_SIZE) * (N/BLOCK_SIZE);
    int threads = BLOCK_SIZE*BLOCK_SIZE;
    block_mm_kernel<<<(blocks,1,1),(threads,1,1)>>>(A,B,C,M,N,BLOCK_SIZE);
	CudaCheckError();
	CudaSafeCall(hipDeviceSynchronize());
	gpu_timer.stop();
	float gpu_time = gpu_timer.getElapsed();
	gpu_timer.end();
	return gpu_time;
}
