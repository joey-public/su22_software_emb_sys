#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "matrixmul.h"
#include "timer.h"

#define BLOCK_SIZE 32 

__global__ void block_mm_kernel(const float* A, const float* B, float* output, int M, int N) 
{
    const int tx=threadIdx.x, ty=threadIdx.y;
    const int bx=blockIdx.x, by=blockIdx.y;
    const int dim=blockDim.x;
    const int row = blockDim.y * by + ty; 
    const int col = blockDim.x * bx + tx; 

    if(row > N || col > N) return;
    __shared__ float aSub[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ float bSub[BLOCK_SIZE*BLOCK_SIZE];
    int cIdx = row*N+col; 
    float cValue = 0.0f;
    for(int i=0; i<M/dim; i++)
    {
       aSub[ty*dim+tx] = A[(row*M) + (i*dim) + tx]; 
       bSub[ty*dim+tx] = B[(i*dim*N) + (ty*N) + col]; 
       __syncthreads();
       for(int j=0;j<BLOCK_SIZE;j++)
       {
            cValue += aSub[ty*dim+j] * bSub[j*dim+tx];
       }
       __syncthreads();
    }
    output[cIdx] = cValue;
}

inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

float run_mm_gpu(const float* A, const float* B, float* C, int M, int N)
{
	Timer gpu_timer;
	gpu_timer.start();
    dim3 gridD(divup(N,BLOCK_SIZE), divup(N,BLOCK_SIZE), 1);
    dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
    block_mm_kernel<<<gridD,blockD>>>(A,B,C,M,N);
	CudaCheckError();
	CudaSafeCall(hipDeviceSynchronize());
	gpu_timer.stop();
	float gpu_time = gpu_timer.getElapsed();
	gpu_timer.end();
	return gpu_time;
}
