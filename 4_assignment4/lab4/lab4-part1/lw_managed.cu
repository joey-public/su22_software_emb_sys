
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#include <string>

__global__ void myKernel(int *m, int *v, int *r){
    r[threadIdx.x]  = v[0] * m[threadIdx.x*3+0];
    r[threadIdx.x] += v[1] * m[threadIdx.x*3+1];
    r[threadIdx.x] += v[2] * m[threadIdx.x*3+2];
}

int main(int argc, char* argv[]){

    uint size_m = 9*sizeof(int);
    uint size_v = 3*sizeof(int);
    int *m, *v, *r;
    hipMallocManaged(&m, size_m);
    hipMallocManaged(&v, size_v);
    hipMallocManaged(&r, size_v);
    
    m[0]=0; m[1]=1; m[2]=2;
    m[3]=2; m[4]=3; m[5]=4;
    m[6]=4; m[7]=5; m[8]=6;

    v[0]=-1; v[1]=0; v[2]=1;

    uint b = 1; // dim3 b(1,1,1);
    uint t = 3; //dim3 t(3,1,1);
    myKernel<<<b,t>>>(m, v, r);
    hipDeviceSynchronize(); 
    int scs = 1;
    for(uint i=0; i<3; i++){
        if(r[i] != 2){
            printf("error! ");
            scs = 0;
        }
        printf("r[%d] = %d\n", i, r[i]);
    }
    if(scs == 1){
        printf("Done with no error\n");
    }
    hipFree(m);
    hipFree(v);
    hipFree(r);
    return 0;
}
